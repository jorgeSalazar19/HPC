#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>


void save_matriz(float *Matrix, int row, int col){
  FILE *f = fopen("result_mult.csv", "a");

  if (f == NULL){
    printf("File error\n");
    exit(-1);
  }

  for (int i = 0; i < row; i++) {
    for (int j = 0; j < col; ++j){
      if(col - 1 == j){
        fprintf(f, "%.2f", Matrix[i * col + j]);
      }
      else{
        fprintf(f, "%.2f, ",  Matrix[i * col + j]);
      }
    }
     fprintf(f, "\n");
  }

  fprintf(f, "\n");
  fclose(f);

  return;
}

__host__
void print(float *M, int rows, int cols){
  printf("\n");
  printf("----------------------------------------\n");
  for(int i = 0; i < rows; i++) {
  		for(int j = 0; j < cols; j++) {
     		printf("%.2f ", M[i * cols + j]);
    	}
		printf("\n");
  }
  printf("----------------------------------------\n");
  printf("\n");
  return;
}


__global__ void matrixMultGPU(float *d_matrix1, float *d_matrix2, float *d_MatrixR, int rowM1 , int rowM2 , int colM1 , int colM2 ) {

	int k = 0;
	float sum = 0.0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;

	if (col < colM2 && row < rowM1) {
		for (k = 0; k < rowM2; k++) {
			sum += d_matrix1[row * colM1 + k] * d_matrix2[k * colM2 + col];
		}
		d_MatrixR[row * colM2 + col] = sum;
		}
	}


__host__
void read_matrix_from_file(float *M, FILE *archivo, int rows, int cols){
	for (int i = 0; i < rows; ++i){
		for (int j = 0; j < cols; ++j){
			fscanf(archivo, "%f,", &M[i * cols + j]);
		}
	}
	fclose(archivo);
	return;
}

bool validate(int colM1 ,int rowM2){

	if (colM1 != rowM2){
		return true;
	}

	else{
		return false;
	}

}


int main(int argc, char** argv){

	if (argc != 3){
		printf("agregue los archivos como parametro\n");
		return 1;
	}

	float *h_matrix1, *h_matrix2, *h_MatrixR;
	int rowM1 , rowM2 , colM1 , colM2;
	hipError_t error = hipSuccess;

	FILE *file_1 , *file_2;
	file_1 = fopen(argv[1], "r");
	file_2 = fopen(argv[2], "r");

	fscanf(file_1, "%d", &rowM1);
	fscanf(file_1, "%d", &colM1);
	fscanf(file_2, "%d", &rowM2);
	fscanf(file_2, "%d", &colM2);

	if (validate(colM1,rowM2)){
		printf("Las matrices son incompatibles y no se pueden multiplicar");
		return 1;
	}

	float sizeM1 = rowM1 * colM1 * sizeof(float);
	float sizeM2 = rowM2 * colM2 * sizeof(float);
	float sizeMR = rowM1 * colM2 * sizeof(float);

	h_matrix1 = (float*)malloc(sizeM1);
	h_matrix2 = (float*)malloc(sizeM2);
	h_MatrixR = (float*)malloc(sizeMR);

	read_matrix_from_file(h_matrix1, file_1, rowM1, colM1);
	read_matrix_from_file(h_matrix2, file_2, rowM2, colM2);

	float *d_matrix1, *d_matrix2, *d_MatrixR;

	error = hipMalloc ((void **) &d_matrix1, sizeM1);
	if (error != hipSuccess){
	printf("Error solicitando memoria en la GPU para d_matrix1\n");
	exit(-1);
}
	error = hipMalloc ((void **) &d_matrix2, sizeM2);
	if (error != hipSuccess){
	printf("Error solicitando memoria en la GPU para d_matrix2\n");
	exit(-1);
}
	error = hipMalloc ((void **) &d_MatrixR, sizeMR);
	if (error != hipSuccess){
	printf("Error solicitando memoria en la GPU para d_MatrixR\n");
	exit(-1);
}
	hipMemcpy(d_matrix1, h_matrix1, sizeM1, hipMemcpyHostToDevice);
	hipMemcpy(d_matrix2, h_matrix2, sizeM2, hipMemcpyHostToDevice);

	dim3 bloques(ceil(colM2/16.0),ceil(rowM1/16.0),1);
	dim3 hilos(16,16,1);

	matrixMultGPU<<<bloques,hilos>>>(d_matrix1,d_matrix2,d_MatrixR, rowM1, rowM2 , colM1 , colM2);

	hipMemcpy(h_MatrixR,d_MatrixR,sizeMR,hipMemcpyDeviceToHost);

	print(h_matrix1, rowM1 , colM1);
	print(h_matrix2, rowM2 , colM2);
	print(h_MatrixR, rowM1 , colM2);
	save_matriz(h_MatrixR , rowM1, colM2);

	hipFree(d_matrix1); hipFree(d_matrix2); hipFree(d_MatrixR);
	free(h_matrix1); free(h_matrix2); free(h_MatrixR);
}
