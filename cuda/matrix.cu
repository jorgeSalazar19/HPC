#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__host__

void fill_vector(float* matrix , size){
	float aux = 2.0;

	for (int i = 0; i < size; ++i)
	{
		matrix[i] = (((float)rand())/(float)(RAND_MAX)) * aux;
	}
}

__global__

void matrixMult(float* d_Matrix , float* d_Result , int width){

	int row = blockIdx.y*blockDim.y+threadIdx.y;

	if(row < width){

		d_Result[row] = d_Matrix[row]*2
	}
}


int main(){

	int n = 1000;

	int width = n * n * sizeof(float)

	float *h_Matrix = (float*) Malloc(width)
	float *h_Result = (float*) Malloc(width)

	fill_vector(h_Matrix, width);

	float *d_Matrix, *d_Result;
	hipMalloc ((void **) &d_Matrix, width)
	hipMalloc ((void **) &d_Result, width)

	hipMemcpy(d_Matrix,h_Matrix,width,hipMemcpyHostToDevice);

	dim3 bloques(ceil(n/20.0),1,1);
	dim3 hilos(10,1,1);

	matrixMult<<<bloques,hilos>>>(d_Matrix,d_Result,width);

	hipMemcpy(h_Result,d_Result,width,hipMemcpyDeviceToHost);

	hipFree(d_Matrix;
	hipFree(d_Result);
	free(h_Matrix:
	free(h_Result);
}
