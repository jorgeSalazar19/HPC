#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>

__host__

void fill_vector(float* matrix , int size){
	float aux = 2.0;

	for (int i = 0; i < size; ++i)
	{
		matrix[i] = (((float)rand())/(float)(RAND_MAX)) * aux;
	}
}

__host__
void print(float *V, int len){
  for (int i = 0; i < len; i++) {
    printf("%.2f ", V[i]);
  }
  printf("\n");
}

__global__

void matrixMult(float* d_Matrix , float* d_Result , int n){

	int row = threadIdx.x + blockDim.x * blockIdx.x;

	if(row < n){
		d_Result[row] = d_Matrix[row] * 2;
	}
}


int main(){

	int n = 100;
	hipError_t error = hipSuccess;

	int width = n * sizeof(float);

	float *h_Matrix = (float*) malloc(width);
	float *h_Result = (float*) malloc(width);

	fill_vector(h_Matrix,n);
	print(h_Matrix,n);

	float *d_Matrix, *d_Result;
	error = hipMalloc ((void **) &d_Matrix, width);
	if (error != hipSuccess){
	printf("Error solicitando memoria en la GPU para d_R\n");
	exit(-1);
}
	error = hipMalloc ((void **) &d_Result, width);
	if (error != hipSuccess){
	printf("Error solicitando memoria en la GPU para d_R\n");
	exit(-1);
}

	hipMemcpy(d_Matrix,h_Matrix,width,hipMemcpyHostToDevice);

	dim3 bloques(ceil(n/10.0),1,1);
	dim3 hilos(10,1,1);

	matrixMult<<<bloques,hilos>>>(d_Matrix,d_Result,n);
	hipDeviceSynchronize();

	hipMemcpy(h_Result,d_Result,width,hipMemcpyDeviceToHost);
	print(h_Result,n);

	hipFree(d_Matrix);
	hipFree(d_Result);
	free(h_Matrix);
	free(h_Result);
}
