#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <malloc.h>

__host__

void fill_vector(float* matrix , int size){
	float aux = 2.0;

	for (int i = 0; i < size; ++i)
	{
		matrix[i] = (((float)rand())/(float)(RAND_MAX)) * aux;
	}
}

__host__
void print(float *V, int len){
  for (int i = 0; i < len; i++) {
    printf("%.2f ", V[i]);
  }
  printf("\n");
}

__global__

void matrixMult(float* d_Matrix , float* d_Result , int width){

	int row = blockIdx.y*blockDim.y+threadIdx.y;

	if(row < width){

		d_Result[row] = d_Matrix[row]*2;
	}
}


int main(){

	int n = 1000;

	int width = n * n * sizeof(float);

	float *h_Matrix = (float*) malloc(width);
	float *h_Result = (float*) malloc(width);

	fill_vector(h_Matrix,n);
	print(h_Matrix,n)

	float *d_Matrix, *d_Result;
	hipMalloc ((void **) &d_Matrix, width);
	hipMalloc ((void **) &d_Result, width);

	hipMemcpy(d_Matrix,h_Matrix,width,hipMemcpyHostToDevice);

	dim3 bloques(ceil(n/20.0),1,1);
	dim3 hilos(10,1,1);

	matrixMult<<<bloques,hilos>>>(d_Matrix,d_Result,width);

	hipMemcpy(h_Result,d_Result,width,hipMemcpyDeviceToHost);

	print(h_Result,n)

	hipFree(d_Matrix);
	hipFree(d_Result);
	free(h_Matrix);
	free(h_Result);
}
